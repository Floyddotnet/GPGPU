#include "hip/hip_runtime.h"
// nvcc -lglut -lGLEW -I/opt/NVIDIA_GPU_Computing_SDK/C/common/inc -L/opt/NVIDIA_GPU_Computing_SDK/C/lib -lcutil_i386 fraktal.cu
// Compile ^^^
// 
// =======================================================================
//    Class: Procesory graficzne w obliczeniach równoległych (CUDA)
//    Task: IFS (list 5)
//    
//    Jan Sochiera, 241745
//    
//    "whitenoise.cu" code reused
//  ----------------------------------------------------------------------  
//    
//    Usage: 
//    	i 	      - zoom in
//    	o 	      - clear and generate default view
//    	a-s and z-x   - modify matrix 
//    	q 	      - quit
//    	any other     - concentrate fractal
//	
//========================================================================
#include <stdio.h>
#include <GL/glew.h>
#include <GL/glut.h>
#include <cutil.h>
#include <cuda_gl_interop.h>
#include <cutil_inline.h>

typedef unsigned int  uint;

#define sieveSize 32 
#define red 255
#define pointsPerThread 48
#define imageWidth 1366
#define imageHeight 742
#define imageH imageHeight
#define imageW imageWidth

float realWidth = 3.31;
float realHeight = 1.8;
float s = 0.97;

__device__ __constant__ float Matrix[12];

//=========================================================================
// Pseudo Random: TausStep, LCGStep, Hybrid based on:
// GPU Gems 3:
// Lee Howes, David Thomas (Imperial College London)
// Chapter 37. Efficient Random Number Generation and Application Using CUDA
//=========================================================================
// Cheap pseudo random numbers:
//  
// S1, S2, S3, M - constants,  z - state
__device__ uint TausStep(uint &z, int S1, int S2, int S3, uint M)  {
    uint b=(((z << S1) ^ z) >> S2);
    return z = (((z & M) << S3) ^ b);
}

// A, C - constants
__device__ uint LCGStep(uint &z, uint A, uint C) {
    return z=(A*z+C);
}

// Mixed :
__device__ float HybridTaus(uint &z1, uint &z2, uint &z3, uint &z4) {
    // Combined period is lcm(p1,p2,p3,p4)~ 2^121
    return 2.3283064365387e-10 * (              // Periods
               TausStep(z1, 13, 19, 12, 4294967294UL) ^   // p1=2^31-1
               TausStep(z2,  2, 25,  4, 4294967288UL) ^   // p2=2^30-1
               TausStep(z3,  3, 11, 17, 4294967280UL) ^   // p3=2^28-1
               LCGStep( z4,    1664525, 1013904223UL)     // p4=2^32
           );
}

// Int Mixed and modified: cheaper
__device__ uint HybridTausInt(uint &z1, uint &z2, uint &z3, uint &z4) {
    // Combined period is lcm(p1,p2,p3,p4)~ 2^121
    return (              // Periods
               TausStep(z1, 13, 19, 12, 4294967294UL) ^   // p1=2^31-1
               //  TausStep(z2,  2, 25,  4, 4294967288UL) ^   // p2=2^30-1
               //  TausStep(z3,  3, 11, 17, 4294967280UL) ^   // p3=2^28-1
               LCGStep( z4,    1664525, 1013904223UL)     // p4=2^32
           );
}

// Testing func:   cheap one int state
__device__ uint funct(uint id) {
    //return LCGStep( id,    1664525, 1013904223UL) ;    // p4=2^32
    return HybridTausInt(id,id,id,id);
    //return id = (1664525*id+1013904223UL) % (65536*256);
    //return id = (xx%256) + 256*(y%256) + 65536*( (256-(xx%256)-(y%256))%256 ) ;
}
//=========================================================================

//=========================================================================
//initialization kernel:
__global__ void initim1(uint * output) {
    uint x  = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint id = __umul24(y, imageWidth) + x;  // Good for < 16MPix

    if ( x < imageWidth && y < imageHeight ) {
	output[id] = 0;
    }
}

//temp to output DeviceToDevice fast(?) copy
__global__ void outputCpy(uint * output, uint * temp) {
    uint x  = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint id = __umul24(y, imageWidth) + x;  // Good for < 16MPix
    if ( x < imageWidth && y < imageHeight ) {
	output[id] = temp[id];
    }
}

#define scale(x, y, fx, fy)\
fx = float(x - (imageW/2)) * realW/(1.0*float(imageW));\
fy = float(y - (imageH/2)) * realH/(1.0*float(imageH));

#define rescale(x, y, fx, fy)\
x = (fx) * float(imageW)/realW + imageW/2;\
y = (fy) * float(imageH)/realH + imageH/2;

//computes and concentrates fractal
__global__ void fractalCompute(uint * output, float realW, float realH, uint seed) {

  int ix = (blockIdx.x * blockDim.x + threadIdx.x);
  int iy = (blockIdx.y * blockDim.y + threadIdx.y);
  uint xSeed  = funct(ix*seed);
  uint ySeed  = funct(iy*seed);
  
  float fx, fy;
  scale(ix, iy, fx, fy);
  
  for(int i = 0; i < pointsPerThread; i++){
    float x = fx;
    float y = fy;
    if(funct(uint(i * seed* 12431 + 327911*xSeed + 98097*ySeed)) % 22519117 > 11500000){
      fx = Matrix[0]*x + Matrix[1]*y + Matrix[2];
      fy = Matrix[3]*x + Matrix[4]*y + Matrix[5];
    }
    else{
      fx = Matrix[6]*x + Matrix[7]*y + Matrix[8];
      fy = Matrix[9]*x + Matrix[10]*y + Matrix[11];
    }
    rescale(ix, iy, fx, fy);
    
    if (i >= sieveSize && abs(fx*2.0) < realW && abs(fy*2.0 ) < realH && ix < imageW && iy < imageH){
      output[iy*imageW + ix] = red;
    }
  }
}



//simply interpolates output and save shrinked view in temp
//calibrated points blackens over time
__global__ void imageScale(uint * output, float realW, float realH, float s, uint *temp) {
  int ix = (blockIdx.x * blockDim.x + threadIdx.x);
  int iy = (blockIdx.y * blockDim.y + threadIdx.y);

  float fx, fy;
  scale(ix, iy, fx, fy);

  int color = 0;
  if (abs(fx*2.0) < realW && abs(fy*2.0 ) < realH && ix < imageW && iy < imageH) color = output[iy*imageW + ix];
  if(color > 5){

    realW *= s;
    realH *= s;
    rescale(ix, iy, fx, fy);
    if (abs(fx*2.0) < realW && abs(fy*2.0 ) < realH && ix < imageW && iy < imageH){
      temp[iy*imageW + ix] = (color - 25);
    }
  }
}


float A[12] = {-0.4, 0, -1, 0, -0.4, 0.1, 0.76, -0.4, 0, 0.4, 0.76, 0};

GLuint   pbo = 0;      // OpenGL PBO id.
uint    *output;   // CUDA device pointer to PBO data
uint 	*tempOutput;

dim3 blockSize(16,16); // threads
dim3 gridSize;         // set up in initPixelBuffer

int iDivUp(int a, int b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void initPixelBuffer() {
    if (pbo) {      // delete old buffer
        cudaGLUnregisterBufferObject(pbo);
        glDeleteBuffersARB(1, &pbo);
    }
    // create pixel buffer object for display
    glGenBuffersARB(1, &pbo);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, imageWidth * imageHeight * sizeof(GLubyte)*4, 0, GL_STREAM_DRAW_ARB);
    cudaGLRegisterBufferObject(pbo);

    // calculate new grid size
    gridSize = dim3(iDivUp(imageWidth, blockSize.x), iDivUp(imageHeight, blockSize.y));

    // from display:
    cudaGLMapBufferObject((void**)&output, pbo  );
    initim1<<<gridSize, blockSize>>>(output);
    CUT_CHECK_ERROR("Kernel error");
    cudaGLUnmapBufferObject(pbo);
    hipMalloc(&tempOutput, imageW*imageH*sizeof(uint));
    initim1<<<gridSize, blockSize>>>(tempOutput);
    CUT_CHECK_ERROR("Kernel error");
}

static int cnt=0; // generation(display calls) count

void display() {
    printf("%4d\n", cnt % 10000);
    cnt++;

    cudaGLMapBufferObject((void**)&output, pbo);
   
    int random = rand();
    
    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer(   timer));
    
    fractalCompute<<<gridSize, blockSize>>>(output, realWidth, realHeight, uint(random));

    hipDeviceSynchronize();
    printf( "GPU time: %2.6f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));
    
    CUT_CHECK_ERROR("Kernel error");
    cudaGLUnmapBufferObject(pbo );

    glClear(GL_COLOR_BUFFER_BIT);
    glDisable(GL_DEPTH_TEST);
    glRasterPos2i(0, 0);
    glDrawPixels(imageWidth, imageHeight, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    glutSwapBuffers();
    glutReportErrors();
}

void reshape(int x, int y) {

    initPixelBuffer();
    glViewport(0, 0, x, y);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0);
}


void keyboard(unsigned char k, int , int ) {
    if (k == 27 || k == 'q' || k == 'Q') exit(1);
    
    if (k == 'i' || k == 'I'){
//      initim1<<<gridSize, blockSize>>>(tempOutput);
//      imageScale<<<gridSize, blockSize>>>(output, realWidth, realHeight, s, tempOutput);  
      realWidth *= s;
      realHeight *= s; 
      initim1<<<gridSize, blockSize>>>(output);  
//      outputCpy<<<gridSize, blockSize>>>(output, tempOutput);  
//      hipMemcpy(output, tempOutput, imageH * imageW * sizeof(uint), hipMemcpyDeviceToDevice);
    }
    
    if (k == 'o' || k == 'O'){
      initim1<<<gridSize, blockSize>>>(output);
      realWidth = 3.2;
      realHeight = 1.8;
      A[3] = 0;
      A[4] = -0.4;
      hipMemcpyToSymbol(HIP_SYMBOL("Matrix"), A, 12*sizeof(*A)); 
    }
    
    if (k == 'z'){
      initim1<<<gridSize, blockSize>>>(output);
      A[3] += 0.01;
      hipMemcpyToSymbol(HIP_SYMBOL("Matrix"), A, 12*sizeof(*A)); 

    }
    
    
    if (k == 'x'){
      initim1<<<gridSize, blockSize>>>(output);
      A[3] -= 0.01;
      hipMemcpyToSymbol(HIP_SYMBOL("Matrix"), A, 12*sizeof(*A)); 
    }
    
    if (k == 'a'){
      initim1<<<gridSize, blockSize>>>(output);
      A[4] += 0.01;
      hipMemcpyToSymbol(HIP_SYMBOL("Matrix"), A, 12*sizeof(*A)); 
    }
    
    
    if (k == 's'){
      initim1<<<gridSize, blockSize>>>(output);
      A[4] -= 0.01;
      hipMemcpyToSymbol(HIP_SYMBOL("Matrix"), A, 12*sizeof(*A));
    }
    
    display();
}

void cleanup() {
    cudaGLUnregisterBufferObject(pbo);
    glDeleteBuffersARB(1, &pbo);
}

int main( int argc, char** argv) {
  
    srand(1337);
    
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
    int windowWidth = imageW;
    int windowHeight = imageH;
    glutInitWindowSize(windowWidth, windowHeight);
    glutCreateWindow("Fractal");
    
    hipMemcpyToSymbol(HIP_SYMBOL("Matrix"), A, 12*sizeof(*A)); 
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutReshapeFunc(reshape);

    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 GL_ARB_pixel_buffer_object")) {
        fprintf(stderr, "OpenGL requirements not fulfilled !!!\n");
        exit(-1);
    }
    initPixelBuffer();

    atexit(cleanup);
    glutMainLoop();
    return 0;
}




















